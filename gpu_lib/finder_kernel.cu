#include "hip/hip_runtime.h"
#include "finder_kernel.cuh"

#include "gpu_memory.h"

namespace {
__device__ uint16_t get_occurrence(uint32_t motif_hash,
                                   uint32_t *seq_hashes,
                                   uint32_t sequences_count,
                                   uint32_t *hash_lengths,
                                   uint32_t *hash_begins,
                                   uint32_t threads_per_block)
{
    // Shared memory size bytes: hash length * sizeof (uint32_t)
    // Max sequence length: shared memory size / 4bytes (16-25k)
    extern __shared__ uint32_t sequence[];

    // Max number of sequences: 64k
    uint16_t result = 0;

    for (uint32_t j = 0; j < sequences_count; j++) {
        uint32_t seq_offset = hash_begins[j];
        uint32_t seq_length = hash_lengths[j];

        // Load sequence to fast shared memory
        for (uint32_t i = threadIdx.x; i < seq_length; i += threads_per_block) {
            if (i < seq_length) {
                sequence[i] = seq_hashes[i + seq_offset];
            }
        }

        __syncthreads();
        // Match motif to sequence
        uint8_t found = 0;
        for (uint32_t i = 0; (i < length) && !found; i++) {
            found = (sequence[i] & motif_hash) == sequence[i];
        }
        result += found;
        __syncthreads();
    }
    return result;
}

 __device__ uint32_t motif_index_to_hash(uint32_t motif_index) {
     uint32_t result = 0;
     uint32_t mult = 1;
     for (uint32_t i = MOTIV_LEN - 1; i >= 0; i--) {
         result += (motif_index % ALPH_SIZE + 1) * mult;
         mult *= HASH_BASE;
         motif_index /= ALPH_SIZE;
     }
     return result;
 }

__global__ void motif_finder_kernel_external(
        uint16_t *weights_out,
        uint32_t *seq_hashes,
        uint32_t sequences_count,
        uint32_t *hash_lengths,
        uint32_t *hash_begins,
        uint32_t *mot_hashes,
        uint32_t mots_to_copy,
        uint32_t threads_per_block)
{
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t motif_hash = (index <= mots_to_copy) ? mot_hashes[index] : 0;
    weights_out[index] = get_occurrence(motif_hash, seq_hashes, sequences_count, hash_lengths, hash_begins, threads_per_block);
}

__global__ void motif_finder_kernel_internal(
        uint16_t *weights_out,
        uint32_t *seq_hashes,
        uint32_t sequences_count,
        uint32_t *hash_lengths,
        uint32_t *hash_begins,
        uint32_t mots_to_copy,
        uint32_t motif_idx_offset,
        uint32_t threads_per_block)
{
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t motif_hash = (index <= mots_to_copy) ? motif_index_to_hash(motif_idx_offset + index) : 0;
    weights_out[index] = get_occurrence(motif_hash, seq_hashes, sequences_count, hash_lengths, hash_begins, threads_per_block);
}
} // namespace


GpuExternalMemory::GpuExternalMemory(const GpuCudaParams &params, const SequenceHashes &sequence_hashes)
{
    allocator = create_memory_allocator(params.unified_memory);

    weights_out = allocate_on_device_and_init<uint16_t>(allocator->get(), params.motif_range_size, nullptr);
    seq_hashes = allocate_on_device_and_init<uint32_t>(allocator->get(), sequence_hashes.hashes.size(), &(sequence_hashes.hashes[0]));
    seq_lengths = allocate_on_device_and_init<uint32_t>(allocator->get(), sequence_hashes.lengths.size(), &(sequence_hashes.lengths[0]));
    seq_begins = allocate_on_device_and_init<uint32_t>(allocator->get(), sequence_hashes.seq_begins.size(), &(sequence_hashes.seq_begins[0]));
    motif_hashes = allocate_on_device_and_init<uint32_t>(allocator->get(), params.motif_range_size, nullptr);
    sequences_count = sequence_hashes.count;
    shared_memory_size = std::max_element(sequence_hashes.lengths.begin(), sequence_hashes.lengths.end()) * sizeof(uint32_t);
    weights_count = params.motif_range_size;
}

GpuExternalMemory::~GpuExternalMemory()
{
    hipFree(weights_out);
    hipFree(seq_hashes);
    hipFree(seq_lengths);
    hipFree(seq_begins);
    hipFree(motif_hashes);
    weights_count = 0;
    sequences_count = 0;
}

void motif_finder_gpu_external(
    const std::vector<uint32_t> &motif_hashes,
    const GpuExternalMemory &mem,
    const GpuCudaParams &params,
    std::vector<uint16_t> &out_motif_weights,
    uint32_t motif_offset,
    uint32_t motifs_count,
    int device_id)
{
    hipSetDevice(device_id);

    mem.allocator->MEMCPY_TO_DEVICE(mem.motif_hashes, &motif_hashes[motif_offset],  motifs_count * sizeof(uint32_t));
    motif_finder_kernel_external<<<params.motif_range_size/params.threads_per_block, params.threads_per_block, mem.shared_memory_size>>>(
        mem.weights_out,
        mem.seq_hashes,
        mem.sequences_count,
        mem.seq_lengths,
        mem.seq_begins,
        mem.motif_hashes,
        motifs_count,
        motif_offset,
        params.threads_per_block);
    hipDeviceSynchronize();
    mem.allocator->MEMCPY_TO_HOST(&(out_motif_weights[motif_offset]), mem.weights_out, motifs_count * sizeof(uint16_t));
}

// INTERNAL

GpuInternalMemory::GpuInternalMemory(const GpuCudaParams &params, const SequenceHashes &sequence_hashes)
{
    allocator = create_memory_allocator(params.unified_memory);

    weights_out = allocate_on_device_and_init<uint16_t>(allocator->get(), params.motif_range_size, nullptr);
    seq_hashes = allocate_on_device_and_init<uint32_t>(allocator->get(), sequence_hashes.hashes.size(), &(sequence_hashes.hashes[0]));
    seq_lengths = allocate_on_device_and_init<uint32_t>(allocator->get(), sequence_hashes.lengths.size(), &(sequence_hashes.lengths[0]));
    seq_begins = allocate_on_device_and_init<uint32_t>(allocator->get(), sequence_hashes.seq_begins.size(), &(sequence_hashes.seq_begins[0]));
    sequences_count = sequence_hashes.count;
    shared_memory_size = std::max_element(sequence_hashes.lengths.begin(), sequence_hashes.lengths.end()) * sizeof(uint32_t);
    weights_count = params.motif_range_size;
}

GpuInternalMemory::~GpuInternalMemory()
{
    hipFree(weights_out);
    hipFree(seq_hashes);
    hipFree(seq_lengths);
    hipFree(seq_begins);
    weights_count = 0;
    sequences_count = 0;
}

void motif_finder_gpu_internal(
    const GpuInternalMemory &mem,
    const GpuCudaParams &params,
    std::vector<uint16_t> &out_motif_weights,
    uint32_t motif_idx_offset,
    uint32_t motifs_count,
    int device_id)
{
    hipSetDevice(device_id);
    motif_finder_kernel_internal<<<params.motif_range_size/params.threads_per_block, params.threads_per_block, mem.shared_memory_size>>>(
        mem.weights_out,
        mem.seq_hashes,
        mem.sequences_count,
        mem.seq_lengths,
        mem.seq_begins,
        motifs_count,
        motif_idx_offset,
        params.threads_per_block);
    hipDeviceSynchronize();
    mem.allocator->MEMCPY_TO_HOST(&(out_motif_weights[motif_idx_offset]), mem.weights_out, motifs_count * sizeof(uint16_t));
}
