#include "hip/hip_runtime.h"
#include "finder_kernel.cuh"

namespace {
__device__ uint16_t get_occurrence(uint32_t motif_hash,
                                   uint32_t *seq_hashes,
                                   uint32_t sequences_count,
                                   uint32_t *hash_lengths,
                                   uint32_t *hash_begins,
                                   uint32_t threads_per_block)
{
    // Shared memory size bytes: hash length * sizeof (uint32_t)
    // Max sequence length: shared memory size / 4bytes (16-25k)
    extern __shared__ uint32_t sequence[];

    // Max number of sequences: 64k
    uint16_t result = 0;

    for (uint32_t j = 0; j < sequences_count; j++) {
        uint32_t seq_offset = hash_begins[j];
        uint32_t seq_length = hash_lengths[j];

        // Load sequence to fast shared memory
        for (uint32_t i = threadIdx.x; i < seq_length; i += threads_per_block) {
            if (i < seq_length) {
                sequence[i] = seq_hashes[i + seq_offset];
            }
        }

        __syncthreads();
        // Match motif to sequence
        uint8_t found = 0;
        for (uint32_t i = 0; (i < length) && !found; i++) {
            found = (sequence[i] & motif_hash) == sequence[i];
        }
        result += found;
        __syncthreads();
    }
    return result;
}

 __device__ uint32_t motif_index_to_hash(uint32_t motif_index) {
     uint32_t result = 0;
     uint32_t mult = 1;
     for (uint32_t i = MOTIV_LEN - 1; i >= 0; i--) {
         result += (motif_index % ALPH_SIZE + 1) * mult;
         mult *= HASH_BASE;
         motif_index /= ALPH_SIZE;
     }
     return result;
 }

__global__ void motif_finder_kernel_external(
        uint16_t *weights_out,
        uint32_t *seq_hashes,
        uint32_t sequences_count,
        uint32_t *hash_lengths,
        uint32_t *hash_begins,
        uint32_t *mot_hashes,
        uint32_t mots_to_copy,
        uint32_t threads_per_block)
{
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t motif_hash = (index <= mots_to_copy) ? mot_hashes[index] : 0;
    weights_out[index] = get_occurrence(motif_hash, seq_hashes, sequences_count, hash_lengths, hash_begins, threads_per_block);
}

__global__ void motif_finder_kernel_internal(
        uint16_t *weights_out,
        uint32_t *seq_hashes,
        uint32_t sequences_count,
        uint32_t *hash_lengths,
        uint32_t *hash_begins,
        uint32_t mots_to_copy,
        uint32_t motif_idx_offset,
        uint32_t threads_per_block)
{
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t motif_hash = (index <= mots_to_copy) ? motif_index_to_hash(motif_idx_offset + index) : 0;
    weights_out[index] = get_occurrence(motif_hash, seq_hashes, sequences_count, hash_lengths, hash_begins, threads_per_block);
}
} // namespace

void motif_finder_gpu_internal(
        uint16_t *weights_out,
        uint32_t *seq_hashes,
        uint32_t sequences_count,
        uint32_t *hash_lengths,
        uint32_t *hash_begins,
        uint32_t mots_to_copy,
        uint32_t motif_range_size,
        uint32_t shared_memory_size,
        uint32_t motif_idx_offset,
        uint32_t threads_per_block)
{
    motif_finder_kernel_internal<<<motif_range_size/threads_per_block, threads_per_block, shared_memory_size>>>(
            weights_out,
            seq_hashes,
            sequences_count,
            hash_lengths,
            hash_begins,
            mots_to_copy,
            motif_idx_offset,
            threads_per_block);
    hipDeviceSynchronize();
}

void motif_finder_gpu_external(
        uint16_t *weights_out,
        uint32_t *seq_hashes,
        uint32_t sequences_count,
        uint32_t *hash_lengths,
        uint32_t *hash_begins,
        uint32_t *motif_hashes,
        uint32_t mots_to_copy,
        uint32_t motif_range_size,
        uint32_t shared_memory_size,
        uint32_t motif_idx_offset,
        uint32_t threads_per_block)
{
    motif_finder_kernel_external<<<motif_range_size/threads_per_block, threads_per_block, shared_memory_size>>>(
            weights_out,
            seq_hashes,
            sequences_count,
            hash_lengths,
            hash_begins,
            motif_hashes,
            mots_to_copy,
            motif_idx_offset,
            threads_per_block);
    hipDeviceSynchronize();
}

