//
// Created by andrey on 06.03.2021.
//

#include "gpu_memory.h"
#include <cstring>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

IGpuMemory::~IGpuMemory() {

}

// unified memory
hipError_t UnifiedGpuMemory::MALLOC(void **devPtr, std::size_t size) {
    return hipMallocManaged(devPtr, size);
}
void *UnifiedGpuMemory::MEMCPY_TO_DEVICE(void *dest, const void *src, std::size_t count) {
    return memcpy((wchar_t *)dest, src, count);
}

void *UnifiedGpuMemory::MEMCPY_TO_HOST(void *dest, const void *src, std::size_t count) {
    return memcpy((wchar_t *)dest, src, count);
}

void *UnifiedGpuMemory::MEMSET(void *ptr, int value, std::size_t num) {
    return memset((wchar_t *)ptr, value, num);
}

// non unified memory
hipError_t NonUnifiedGpuMemory::MALLOC(void **devPtr, std::size_t size) {
    return hipMalloc(devPtr, size);
}

void *NonUnifiedGpuMemory::MEMCPY_TO_DEVICE(void *dest, const void *src, std::size_t count) {
    hipMemcpy(dest, src, count, hipMemcpyHostToDevice);
    return dest;
}

void *NonUnifiedGpuMemory::MEMCPY_TO_HOST(void *dest, const void *src, std::size_t count) {
    hipMemcpy(dest, src, count, hipMemcpyDeviceToHost);
    return dest;
}

void *NonUnifiedGpuMemory::MEMSET(void *ptr, int value, std::size_t num) {
    hipMemset(ptr, value, num);
    return ptr;
}

// factory
GpuMemoryPtr create_memory_allocator(bool unified) {
    if (unified) {
        return GpuMemoryPtr(new UnifiedGpuMemory());
    } else {
        return GpuMemoryPtr(new NonUnifiedGpuMemory());
    }
}
